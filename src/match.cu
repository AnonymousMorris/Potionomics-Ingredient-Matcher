#include "hip/hip_runtime.h"
// #pragma GCC optimize("O3","unroll-loops")
// #pragma GCC target("avx","avx2")
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Ingredient.h"

using namespace std;

/* We store ingredient magniums in ingredient_offsets. The first ingredient takes up
ingredient_offsets[0..5). The second one takes up ingredient_offsets[5..10), etc. */
__constant__ int ingredient_offsets[1024];

__global__ void calc_dp(int* dp, int i, int max_ingredients, int n_states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_states) {
        return;
    }
    int offset = ingredient_offsets[i];
    int tmp = dp[idx + (i - 1) * n_states];
    for (int ii = 1; ii <= max_ingredients; ii++) {
        int prev = idx - offset * ii;
        if (prev < 0) {
            break;
        }
        if (dp[prev + (i - 1) * n_states] > 0 || prev == 0) {
            tmp = min(tmp, dp[prev + (i - 1) * n_states] + ii);
        }
    }
    dp[idx + i * n_states] = tmp;
}

__global__ void calc_dp_first(int* dp, int n_states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_states) {
        return;
    }
    int offset = ingredient_offsets[0];
    if (idx % offset == 0) {
        int dp_idx = idx / offset;
        dp[idx] = dp_idx;
    }
}

__global__ void get_recipe(int* dp, int n_ingredients, int n_states, int* result, int* result_n) {
    result_n = 0;
    int idx = n_ingredients * n_states - 1;
    for (int i = n_ingredients - 1; i > 0; i--) {
        int ing_offset = ingredient_offsets[i];
        if (ing_offset <= idx % n_states) {
            if (dp[idx - ing_offset] == dp[idx] - 1) {
                result[*result_n++] = i;
            }
        }
    }
    if (idx == ingredient_offsets[0]) {
        result[*result_n++] = 0;
    }
}

int calc_index(Ingredient ingredient, Ingredient mx_mags) {
    int val = 0;
    for (int i = 0, base = 1; i < 5; i++) {
        val += base * ingredient[i];
        base *= (mx_mags[i] + 1);
    }
    return val;
}

void cuda_calculate_recipe(
    vector<Ingredient> ingredients,  // a vector of ingredients
    int ingredient_limit,            // the maximum number of ingredients allowed
    array<int, 5> maximum_magniums   // maximum maginums the dp will try to get
) {
    int max_states = 1;
    for (int i : maximum_magniums) {
        max_states *= i;
    }
    int num_ingredients = ingredients.size();
    assert(num_ingredients < 1024);

    // Need to prepare ingredient data for copying to constant memory
    // This assumes you have a way to extract offsets from ingredients
    int h_ingredient_offsets[1024];
    // Fill h_ingredient_offsets with data from ingredients vector
    // ... (add your logic here)

    hipMemcpyToSymbol(HIP_SYMBOL(ingredient_offsets), h_ingredient_offsets, sizeof(int) * num_ingredients);

    int* dp;
    int size = num_ingredients * max_states;
    hipMalloc((void**)&dp, sizeof(int) * size);
    hipMemset(dp, 0, sizeof(int) * size);

    calc_dp_first<<<dim3((max_states + 1023) / 1024, 1, 1), dim3(1024, 1, 1)>>>(dp, max_states);

    for (int i = 1; i < num_ingredients; i++) {
        calc_dp<<<dim3((max_states + 1023) / 1024, 1, 1), dim3(1024, 1, 1)>>>(
            dp, i, ingredient_limit, max_states);
    }

    int* h_result = (int*)malloc(num_ingredients * sizeof(int));
    int h_result_n;

    int* d_result;
    int* d_result_n;
    hipMalloc((void**)&d_result, num_ingredients * sizeof(int));
    hipMalloc((void**)&d_result_n, sizeof(int));
    get_recipe<<<dim3(1, 1, 1), dim3(1, 1, 1)>>>(dp, num_ingredients, max_states, d_result,
                                                 d_result_n);
    hipDeviceSynchronize();
    hipMemcpy(h_result, &d_result, num_ingredients, hipMemcpyDeviceToHost);
    hipMemcpy(h_result_n, &d_result_n, 1, hipMemcpyDeviceToHost);

    hipFree(dp);
}
